#include "hip/hip_runtime.h"
#pragma warning(disable : 4201)

#ifndef NDEBUG
    #include <stdio.h>
    #include <stdlib.h>
    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
        if (code != hipSuccess) {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }
    #define INLINE __device__ __host__
#else
    #define INLINE __device__ __host__ __forceinline__
    #define gpuErrchk(ans) (ans);
#endif

#define checkErrors() gpuErrchk(hipPeekAtLastError())
#define uploadNto(cpu_ptr, gpu_ptr, N, offset) gpuErrchk(hipMemcpy(&((gpu_ptr)[(offset)]), (cpu_ptr), sizeof((cpu_ptr)[0]) * (N), hipMemcpyHostToDevice))
#define uploadN(  cpu_ptr, gpu_ptr, N        ) gpuErrchk(hipMemcpy(&((gpu_ptr)[0])       , (cpu_ptr), sizeof((cpu_ptr)[0]) * (N), hipMemcpyHostToDevice))
#define downloadN(gpu_ptr, cpu_ptr, N)         gpuErrchk(hipMemcpyFromSymbol(cpu_ptr     , HIP_SYMBOL((gpu_ptr)), sizeof((cpu_ptr)[0]) * (N), 0, hipMemcpyDeviceToHost))

#define USE_GPU_DEFAULT true

#include "./demo_scene.c"

__device__   u32 d_pixels[MAX_WIDTH * MAX_HEIGHT];

PointLight *d_point_lights;
QuadLight  *d_quad_lights;
Material   *d_materials;
Primitive  *d_primitives;
Mesh       *d_meshes;
Triangle   *d_triangles;
u32        *d_scene_bvh_leaf_ids;
BVHNode    *d_scene_bvh_nodes;
BVHNode    *d_mesh_bvh_nodes;

u32 *d_mesh_bvh_node_counts,
    *d_mesh_triangle_counts;

__global__ void d_render(ProjectionPlane projection_plane, enum RenderMode mode, vec3 camera_position, Trace trace,
                         u16 width,
                         u32 pixel_count,

                         Scene scene,
                         u32        *scene_bvh_leaf_ids,
                         BVHNode    *scene_bvh_nodes,
                         BVHNode    *mesh_bvh_nodes,
                         Mesh       *meshes,
                         Triangle   *mesh_triangles,
                         PointLight *point_lights,
                         QuadLight  *quad_lights,
                         Material   *materials,
                         Primitive  *primitives,

                         const u32 *mesh_bvh_node_counts,
                         const u32 *mesh_triangle_counts
) {
    u32 i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= pixel_count)
        return;

    Pixel *pixel = (Pixel *)&d_pixels[i];

    u16 x = i % width;
    u16 y = i / width;

    Ray ray;
    ray.origin = camera_position;
    ray.direction = normVec3(scaleAddVec3(projection_plane.down, y, scaleAddVec3(projection_plane.right, x, projection_plane.start)));

    scene.point_lights = point_lights;
    scene.quad_lights  = quad_lights;
    scene.materials    = materials;
    scene.primitives   = primitives;
    scene.meshes       = meshes;
    scene.bvh.nodes    = scene_bvh_nodes;
    scene.bvh.leaf_ids = scene_bvh_leaf_ids;

    u32 scene_stack[6], mesh_stack[5];
    trace.mesh_stack  = mesh_stack;
    trace.scene_stack = scene_stack;

    Mesh *mesh = meshes;
    u32 nodes_offset = 0;
    u32 triangles_offset = 0;
    for (u32 m = 0; m < scene.settings.meshes; m++, mesh++) {
        mesh->bvh.node_count = mesh_bvh_node_counts[m];
        mesh->triangle_count = mesh_triangle_counts[m];
        mesh->normals_count  = mesh_triangle_counts[m];
        mesh->triangles      = mesh_triangles + triangles_offset;
        mesh->bvh.nodes      = mesh_bvh_nodes + nodes_offset;

        nodes_offset        += mesh->bvh.node_count;
        triangles_offset    += mesh->triangle_count;
    }

    ray.direction_reciprocal = oneOverVec3(ray.direction);
    trace.closest_hit.distance = trace.closest_hit.distance_squared = INFINITY;

    rayTrace(&ray, &trace, &scene, mode, x, y, pixel);
}

void renderOnGPU(Scene *scene, Viewport *viewport) {
    setViewportProjectionPlane(viewport);

    Dimensions *dim = &viewport->frame_buffer->dimensions;
    u32 pixel_count = dim->width_times_height;
    u16 threads = 256;
    u16 blocks  = pixel_count / threads;
    if (pixel_count < threads) {
        threads = pixel_count;
        blocks = 1;
    } else if (pixel_count % threads)
        blocks++;

    d_render<<<blocks, threads>>>(
            viewport->projection_plane, viewport->settings.render_mode, viewport->camera->transform.position, viewport->trace,

            dim->width,
            pixel_count,

            *scene,
            d_scene_bvh_leaf_ids,
            d_scene_bvh_nodes,
            d_mesh_bvh_nodes,
            d_meshes,
            d_triangles,
            d_point_lights,
            d_quad_lights,
            d_materials,
            d_primitives,

            d_mesh_bvh_node_counts,
            d_mesh_triangle_counts);

    checkErrors()
    downloadN(d_pixels, (u32*)viewport->frame_buffer->pixels, dim->width_times_height)
}

void allocateDeviceScene(Scene *scene) {
    u32 total_triangles = 0;
    if (scene->settings.point_lights) gpuErrchk(hipMalloc(&d_point_lights, sizeof(PointLight) * scene->settings.point_lights))
    if (scene->settings.quad_lights)  gpuErrchk(hipMalloc(&d_quad_lights,  sizeof(QuadLight)  * scene->settings.quad_lights))
    if (scene->settings.primitives)   gpuErrchk(hipMalloc(&d_primitives,   sizeof(Primitive)  * scene->settings.primitives))
    if (scene->settings.meshes) {
        for (u32 i = 0; i < scene->settings.meshes; i++)
            total_triangles += scene->meshes[i].triangle_count;

        gpuErrchk(hipMalloc(&d_meshes,    sizeof(Mesh)     * scene->settings.meshes))
        gpuErrchk(hipMalloc(&d_triangles, sizeof(Triangle) * total_triangles))

        gpuErrchk(hipMalloc(&d_mesh_bvh_node_counts, sizeof(u32) * scene->settings.meshes))
        gpuErrchk(hipMalloc(&d_mesh_triangle_counts, sizeof(u32) * scene->settings.meshes))
    }

    gpuErrchk(hipMalloc(&d_materials,          sizeof(Material) * scene->settings.materials))
    gpuErrchk(hipMalloc(&d_scene_bvh_leaf_ids, sizeof(u32)      * scene->settings.primitives))
    gpuErrchk(hipMalloc(&d_scene_bvh_nodes,    sizeof(BVHNode)  * scene->settings.primitives * 2))
    gpuErrchk(hipMalloc(&d_mesh_bvh_nodes,     sizeof(BVHNode)  * total_triangles * 2))
}

void uploadPrimitives(Scene *scene) {
    uploadN(scene->primitives, d_primitives, scene->settings.primitives)
}

void uploadLights(Scene *scene) {
    if (scene->settings.point_lights) uploadN( scene->point_lights, d_point_lights, scene->settings.point_lights)
    if (scene->settings.quad_lights)  uploadN( scene->quad_lights,  d_quad_lights,  scene->settings.quad_lights)
}

void uploadScene(Scene *scene) {
    uploadLights(scene);
    uploadPrimitives(scene);
    uploadN( scene->materials, d_materials,scene->settings.materials)
}

void uploadSceneBVH(Scene *scene) {
    uploadN(scene->bvh.nodes,    d_scene_bvh_nodes,   scene->bvh.node_count)
    uploadN(scene->bvh.leaf_ids, d_scene_bvh_leaf_ids,scene->settings.primitives)
}

void uploadMeshBVHs(Scene *scene) {
    Mesh *mesh = scene->meshes;
    u32 nodes_offset = 0;
    u32 triangles_offset = 0;
    for (u32 i = 0; i < scene->settings.meshes; i++, mesh++) {
        uploadNto(mesh->bvh.nodes, d_mesh_bvh_nodes, mesh->bvh.node_count, nodes_offset)
        uploadNto(mesh->triangles, d_triangles,      mesh->triangle_count, triangles_offset)
        nodes_offset        += mesh->bvh.node_count;
        triangles_offset    += mesh->triangle_count;
    }

    uploadN(scene->mesh_bvh_node_counts, d_mesh_bvh_node_counts, scene->settings.meshes)
    uploadN(scene->mesh_triangle_counts, d_mesh_triangle_counts, scene->settings.meshes)
}